#include "hip/hip_runtime.h"
#include <stdio.h>
#include "simpleMPI.h"

static void HandleError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit (EXIT_FAILURE);
	}
}
#define CUDA_CHECK( err ) (HandleError( err, __FILE__, __LINE__ ))


// Device code
// Very simple GPU Kernel that computes square roots of input numbers
__global__ void simpleMPIKernel(float *input, float *output)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    output[tid] = sqrt(input[tid]);
}


// Initialize an array with random data (between 0 and 1)
extern "C" void initData(float *data, int dataSize)
{
    for (int i = 0; i < dataSize; i++)
    {
        data[i] = (float)rand() / RAND_MAX;
    }
}

// CUDA computation on each node
// No MPI here, only CUDA
extern "C" void computeGPU(float *hostData, int blockSize, int gridSize)
{
    int dataSize = blockSize * gridSize;

    // Allocate data on GPU memory
    float *deviceInputData = NULL;
    CUDA_CHECK(hipMalloc((void **)&deviceInputData, dataSize * sizeof(float)));

    float *deviceOutputData = NULL;
    CUDA_CHECK(hipMalloc((void **)&deviceOutputData, dataSize * sizeof(float)));

    // Copy to GPU memory
    CUDA_CHECK(hipMemcpy(deviceInputData, hostData, dataSize * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    simpleMPIKernel<<<gridSize, blockSize>>>(deviceInputData, deviceOutputData);

    // Copy data back to CPU memory
    CUDA_CHECK(hipMemcpy(hostData, deviceOutputData, dataSize *sizeof(float), hipMemcpyDeviceToHost));

    // Free GPU memory
    CUDA_CHECK(hipFree(deviceInputData));
    CUDA_CHECK(hipFree(deviceOutputData));
}

float sum(float *data, int size)
{
    float accum = 0.f;

    for (int i = 0; i < size; i++)
    {
        accum += data[i];
    }

    return accum;
}
