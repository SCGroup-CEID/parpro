#include "hip/hip_runtime.h"
/*
 *
 * stream2Async.cu
 *
 * Microbenchmark to illustrate a bandwidth-limited workload.
 *
 * It separately measures the host->device transfer time, kernel
 * processing time, and device->host transfer time.  Due to low
 * arithmetic density in the saxpyGPU() kernel, the bulk of time
 * is spent transferring data. 
 *
 * Build with: nvcc -I ../chLib stream2Async.cu
 *
 * Copyright (c) 2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <chError.h>
#include <chCommandLine.h>
#include <chTimer.h>

#include <stdio.h>
#include <stdlib.h>

#include "saxpyCPU.h"
#include "saxpyGPU.cuh"

hipError_t
MeasureTimes( 
    float *msTotal,
    float *msWallClock,
    float *msHtoD, 
    float *msKernel, 
    float *msDtoH, 
    size_t N, 
    float alpha,
    int nBlocks, 
    int nThreads )
{
    hipError_t status;
    chTimerTimestamp chStart, chStop;
    float *dptrOut = 0, *hptrOut = 0;
    float *dptrY = 0, *hptrY = 0;
    float *dptrX = 0, *hptrX = 0;
    hipEvent_t evStart = 0;
    hipEvent_t evHtoD = 0;
    hipEvent_t evKernel = 0;
    hipEvent_t evDtoH = 0;

    CUDART_CHECK( hipHostAlloc( &hptrOut, N*sizeof(float), 0 ) );
    memset( hptrOut, 0, N*sizeof(float) );
    CUDART_CHECK( hipHostAlloc( &hptrY, N*sizeof(float), 0 ) );
    CUDART_CHECK( hipHostAlloc( &hptrX, N*sizeof(float), 0 ) );

    CUDART_CHECK( hipMalloc( &dptrOut, N*sizeof(float) ) );
    CUDART_CHECK( hipMemset( dptrOut, 0, N*sizeof(float) ) );

    CUDART_CHECK( hipMalloc( &dptrY, N*sizeof(float) ) );
    CUDART_CHECK( hipMemset( dptrY, 0, N*sizeof(float) ) );

    CUDART_CHECK( hipMalloc( &dptrX, N*sizeof(float) ) );
    CUDART_CHECK( hipMemset( dptrY, 0, N*sizeof(float) ) );

    CUDART_CHECK( hipEventCreate( &evStart ) );
    CUDART_CHECK( hipEventCreate( &evHtoD ) );
    CUDART_CHECK( hipEventCreate( &evKernel ) );
    CUDART_CHECK( hipEventCreate( &evDtoH ) );
    for ( size_t i = 0; i < N; i++ ) {
        hptrX[i] = (float) rand() / RAND_MAX;
        hptrY[i] = (float) rand() / RAND_MAX;
    }

    //
    // begin timing
    //

    chTimerGetTime( &chStart );
    CUDART_CHECK( hipEventRecord( evStart, 0 ) );
        CUDART_CHECK( hipMemcpyAsync( dptrX, hptrX, N*sizeof(float), hipMemcpyHostToDevice, NULL ) );
        CUDART_CHECK( hipMemcpyAsync( dptrY, hptrY, N*sizeof(float), hipMemcpyHostToDevice, NULL ) );
    CUDART_CHECK( hipEventRecord( evHtoD, 0 ) );
        saxpyGPU<<<nBlocks, nThreads>>>( dptrOut, dptrX, dptrY, N, alpha );
    CUDART_CHECK( hipEventRecord( evKernel, 0 ) );
        CUDART_CHECK( hipMemcpyAsync( hptrOut, dptrOut, N*sizeof(float), hipMemcpyDeviceToHost, NULL ) );
    CUDART_CHECK( hipEventRecord( evDtoH, 0 ) );
    CUDART_CHECK( hipDeviceSynchronize() );
    chTimerGetTime( &chStop );
    *msWallClock = 1000.0f*chTimerElapsedTime( &chStart, &chStop );

    //
    // end timing
    //

    for ( size_t i = 0; i < N; i++ ) {
        if ( fabsf( hptrOut[i] - (alpha*hptrX[i]+hptrY[i]) ) > 1e-5f ) {
            status = hipErrorUnknown;
            goto Error;
        }
    }
    CUDART_CHECK( hipEventElapsedTime( msHtoD, evStart, evHtoD ) );
    CUDART_CHECK( hipEventElapsedTime( msKernel, evHtoD, evKernel ) );
    CUDART_CHECK( hipEventElapsedTime( msDtoH, evKernel, evDtoH ) );
    CUDART_CHECK( hipEventElapsedTime( msTotal, evStart, evDtoH ) );
Error:
    hipEventDestroy( evDtoH );
    hipEventDestroy( evKernel );
    hipEventDestroy( evHtoD );
    hipEventDestroy( evStart );
    hipFree( dptrOut );
    hipFree( dptrX );
    hipFree( dptrY );
    hipHostFree( hptrOut );
    hipHostFree( hptrX );
    hipHostFree( hptrY );
    return status;
}

double
Bandwidth( float ms, double NumBytes )
{
    return NumBytes / (1000.0*ms);
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    int N_Mfloats = 128;
    size_t N;
    int nBlocks = 1500;
    int nThreads = 256;
    float alpha = 2.0f;

    chCommandLineGet( &nBlocks, "nBlocks", argc, argv );
    chCommandLineGet( &nThreads, "nThreads", argc, argv );
    chCommandLineGet( &N_Mfloats, "N", argc, argv );
    printf( "Measuring times with %dM floats", N_Mfloats );
    if ( N_Mfloats==128 ) {
        printf( " (use --N to specify number of Mfloats)");
    }
    printf( "\n" );

    N = 1048576*N_Mfloats;

    CUDART_CHECK( hipSetDeviceFlags( hipDeviceMapHost ) );
    {
        float msTotal, msWallClock, msHtoD, msKernel, msDtoH;
        CUDART_CHECK( MeasureTimes( &msTotal, &msWallClock, &msHtoD, &msKernel, &msDtoH, N, alpha, nBlocks, nThreads ) );
        printf( "Memcpy( host->device ): %.2f ms (%.2f MB/s)\n", msHtoD, Bandwidth( msHtoD, 2*N*sizeof(float) ) );
        printf( "Kernel processing     : %.2f ms (%.2f MB/s)\n", msKernel, Bandwidth( msKernel, 3*N*sizeof(float) ) );
        printf( "Memcpy (device->host ): %.2f ms (%.2f MB/s)\n\n", msDtoH, Bandwidth( msDtoH, N*sizeof(float) ) );
        printf( "Total time (wall clock): %.2f ms (%.2f MB/s)\n", msWallClock, Bandwidth( msWallClock, 3*N*sizeof(float) ) );
    }

Error:
    if ( status == hipErrorOutOfMemory ) {
        printf( "Memory allocation failed\n" );
    }
    else if ( hipSuccess != status ) {
        printf( "Failed\n" );
    }
    return hipSuccess != status;
}
